#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/video/tracking.hpp>
#include "stdafx.h"

typedef unsigned char uchar;
typedef unsigned int uint;

#define BLOCK_SIZE 32

int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__
int Signe(int pValue)
{
	int ret = 0;
	if (pValue >= 0)
		ret = 1;
	return ret;
}

// kernel pour l'op�rateur LBP
__global__
void Kernel_LBP(uchar* imgIn, uchar* imgOut, int ImgWidth, int imgHeight) {
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;

	int Index = (ImgNumLigne * ImgWidth) + ImgNumColonne;

	if ((ImgNumColonne < ImgWidth - 2) && (ImgNumLigne < imgHeight - 2)) { // ne pas calculer les bordures
		// valeur de centre
		int centerValue = imgIn[Index + ImgWidth + 1];
		
		// signe des valeurs
		int tl = Signe(imgIn[Index] - centerValue);
		int tc = Signe(imgIn[Index + 1] - centerValue);
		int tr = Signe(imgIn[Index + 2] - centerValue);
		int cl = Signe(imgIn[Index + ImgWidth] - centerValue);
		int cr = Signe(imgIn[Index + ImgWidth + 2] - centerValue);
		int bl = Signe(imgIn[Index + ImgWidth * 2] - centerValue);
		int bc = Signe(imgIn[Index + ImgWidth * 2 + 1] - centerValue);
		int br = Signe(imgIn[Index + ImgWidth * 2 + 2] - centerValue);

		imgOut[Index] = tl * 1 + tc * 2 + tr * 4 + cl * 8 + cr * 16 + bl * 32 + bc * 64 + bc * 128;
	}
}

extern "C" bool GPGPU_LBP(cv::Mat* imgIn, cv::Mat* imgOut)
{
	//	1. Initialize data
	hipError_t cudaStatus;
	uchar* gDevImage;
	uchar* gDevImageOut;

	uint imageSize = imgIn->rows * imgIn->step1();

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgIn->cols, BLOCK_SIZE), iDivUp(imgIn->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc(&gDevImageOut, imageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgIn->data, imageSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//	4. Launch kernel
	Kernel_LBP << <dimGrid, dimBlock >> >(gDevImage, gDevImageOut, imgIn->cols, imgIn->rows);

	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(imgOut->data, gDevImageOut, imageSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//	6. Free GPU memory
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return cudaStatus;
}